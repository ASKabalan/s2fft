#include <algorithm>
#include "s2fft.h"
#include <array>
#include <complex>
#include <iostream>
#include <ostream>
#include <sstream>
#include "hipfft/hipfft.h"
#include "perfostep.hpp"
#include "s2fft_kernels.h"
#include <argparse.hpp>

using namespace s2fft;

enum SpectralDirection { EXTENSION, FOLDING };

void run_test(int nside, std::string type, int L, bool print_res) {
    int total_pixels = 12 * nside * nside;

    // Compute the flm size
    int polar_pixels = 4 * nside * (nside - 1);
    int equator_rings_num = (total_pixels - polar_pixels) / (4 * nside);
    int num_rings = 4 * nside - 1;
    int flm_size = num_rings * (2 * L);

    SpectralDirection dir = (type == "extended") ? EXTENSION : FOLDING;

    int input_size = dir == EXTENSION ? total_pixels : flm_size;
    int output_size = dir == EXTENSION ? flm_size : total_pixels;

    std::cout << "input_size: " << input_size << std::endl;
    std::cout << "output_size: " << output_size << std::endl;
    std::cout << "Direction: " << (dir == EXTENSION ? "EXTENSION" : "FOLDING") << std::endl;

    int *h_vec_in = new int[input_size];
    int *h_vec_out = new int[output_size];
    int *d_vec_in;
    int *d_vec_out;

    hipMalloc(&d_vec_in, input_size * sizeof(int));
    hipMalloc(&d_vec_out, output_size * sizeof(int));

    // Initialize host vectors using std::generate
    int start_index(0);
    std::generate(h_vec_in, h_vec_in + input_size, [&start_index]() {
        int c;
        c = start_index;
        start_index += 1;
        return c;
    });

    // Copy host data to device
    checkCudaErrors(hipMemcpy(d_vec_in, h_vec_in, input_size * sizeof(int), hipMemcpyHostToDevice));

    ////// Create cudastream
    hipStream_t stream;
    hipStreamCreate(&stream);

    s2fftDescriptor desc(nside, L, true, true, fft_norm::BACKWARD, false);
    s2fftExec exec;
    size_t worksize(0);
    exec.Initialize(desc, worksize);

    if (dir == EXTENSION) {
        s2fftKernels::launch_spectral_extension(d_vec_in, d_vec_out, nside, L, exec.m_equatorial_offset_start,
                                                exec.m_equatorial_offset_end, stream);
    } else {
        s2fftKernels::launch_spectral_folding(d_vec_in, d_vec_out, nside, L, exec.m_equatorial_offset_start,
                                              exec.m_equatorial_offset_end, stream);
    }

    hipStreamSynchronize(stream);
    checkCudaErrors(hipGetLastError());

    // Copy device data to host
    checkCudaErrors(hipMemcpy(h_vec_out, d_vec_out, output_size * sizeof(int), hipMemcpyDeviceToHost));

    // Print results
    for (int i = 0; i < output_size; i++) {
        std::cout << "[" << i << "] " << h_vec_out[i] << std::endl;
    }

    // Free memory
    delete[] h_vec_in;
    delete[] h_vec_out;
    hipFree(d_vec_in);
    hipFree(d_vec_out);
}

int main(int argc, char **argv) {
    argparse::ArgumentParser program("s2fft_kernels_test");
    program.add_argument("-s", "--nside").help("nside").scan<'i', int>();
    program.add_argument("-t", "--type")
            .help("Spectral type : extended folded or both")
            .default_value("extended");
    program.add_argument("-L", "--lmax").help("lmax").default_value(2).scan<'i', int>();
    program.add_argument("-p", "--print")
            .help("Print results [Boolean]")
            .default_value(false)
            .implicit_value(true);

    try {
        program.parse_args(argc, argv);
    } catch (const std::runtime_error &err) {
        std::cout << err.what() << std::endl;
        std::cout << program;
        exit(1);
    }
    int nside = program.get<int>("--nside");
    std::string type = program.get<std::string>("--type");
    int L = program.get<int>("--lmax");
    bool print_res = program.get<bool>("--print");

    run_test(nside, type, L, print_res);

    return 0;
}
