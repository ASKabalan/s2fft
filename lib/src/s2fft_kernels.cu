#include "hip/hip_runtime.h"
#include "s2fft_kernels.h"
#include "hresult.h"
#include <cmath>  // has to be included before cuda/std/complex
#include <cstddef>
#include <cuda/std/complex>

namespace s2fftKernels {

__global__ void spectral_extension(hipfftComplex* data, hipfftComplex* output, int nside, int L,
                                   int equatorial_ring_num) {
    // Copy data into registers
    int _nside = nside;
    int _L = L;
    int _equatorial_ring_num = equatorial_ring_num;
    // Kernel is launched with a block size of flm_size
    // the ftm size is 2 * L
    // In this case the block size is 2 * L and the blockId is the ring number

    // Compute nphi of current ring
    int nphi(0);

    // Upper Polar rings
    if (blockIdx.x < _nside - 1) {
        nphi = 4 * (blockIdx.x + 1);

    }
    // Lower Polar rings
    else if (blockIdx.x < 2 * _nside + _equatorial_ring_num - 1) {
        nphi = 4 * (_nside - (blockIdx.x - _nside));
    }
    // Equatorial ring
    else {
        nphi = 4 * _nside;
    }

    extern __shared__ hipfftComplex ring[];

    // Load the ring into shared memory
    if (threadIdx.x > (_L - nphi) / 2 && threadIdx.x < (_L + nphi) / 2) {
        // Data is guaranteed to be accessed without data races (no need for atomic operations)
        ring[threadIdx.x - (_L - nphi) / 2] = data[blockIdx.x * _L + threadIdx.x];
    }

    __syncthreads();

    // Spectral extension
    // The resulting array has size 2 * L and it has these indices :

    // fm[-jnp.arange(L - nphi // 2, 0, -1) % nphi],
    // fm,
    // fm[jnp.arange(L - (nphi + 1) // 2) % nphi],

    // The first part of the array is the negative part of the spectrum
    // The second part is the original spectrum
    // The third part is the positive part of the spectrum

    // Compute the negative part of the spectrum
    if (threadIdx.x < (_L - nphi) / 2) {
        int index = (threadIdx.x + 1) % nphi;
        index = nphi - index;
        output[blockIdx.x * _L + threadIdx.x] = ring[index];
    }
    // Compute the central part of the spectrum
    else if (threadIdx.x >= (_L - nphi) / 2 && threadIdx.x < (_L + nphi) / 2) {
        output[blockIdx.x * _L + threadIdx.x] = ring[threadIdx.x - (_L - nphi) / 2];
    }
    // Compute the positive part of the spectrum
    else {
        int index = (threadIdx.x - (_L + nphi) / 2) % nphi;
        output[blockIdx.x * _L + threadIdx.x] = ring[index];
    }
}

HRESULT launch_spectral_extension(hipfftComplex* data, hipfftComplex* output, const int& nside, const int& L,
                                  const int& equatorial_ring_num, hipStream_t stream) {
    // Launch the kernel

    int block_size = 2 * L;
    int grid_size = 2 * nside + equatorial_ring_num;

    spectral_extension<<<grid_size, block_size, block_size * sizeof(hipfftComplex), stream>>>(
            data, output, nside, L, equatorial_ring_num);

    checkCudaErrors(hipGetLastError());
    return S_OK;
}

}  // namespace s2fftKernels