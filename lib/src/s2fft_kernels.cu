#include "hip/hip_runtime.h"
#include "s2fft_kernels.h"
#include "hresult.h"
#include <cmath>  // has to be included before cuda/std/complex
#include <cstddef>
#include <cuda/std/complex>
#include <iostream>

namespace s2fftKernels {

__global__ void spectral_folding(int* data, int* output, int nside, int L, int equatorial_offset_start,
                                 int equatorial_offset_end) {
    // few inits
    int polar_rings = nside - 1;
    int equator_rings = 3 * nside + 1;
    int total_rings = 4 * nside - 1;
    int ftm_size = 2 * L;
    // Compute number of pixels
    int total_pixels = 12 * nside * nside;
    int upper_pixels = nside * (nside - 1) * 2;
    int equator_pixels = 4 * nside * equator_rings;
    // Which ring are we working on
    int current_indx = blockIdx.x * blockDim.x + threadIdx.x;
    int pos(0);
    int indx(0);
    // Compute nphi of current ring
    int nphi(0);

    // ring index
    int ring_index = current_indx / (2 * L);
    // offset for the FTM slice
    int offset = current_indx % (2 * L);
    // offset for original healpix ring
    // Sum of all elements from 0 to n is  n * (n + 1) / 2 in o(1) time .. times 4 to get the number of
    // elements before current ring
    int offset_ring(0);

    // Upper Polar rings
    if (ring_index < nside - 1) {
        nphi = 4 * (ring_index + 1);
        offset_ring = ring_index * (ring_index + 1) * 2;
        pos = 1;
    }
    // Lower Polar rings
    else if (ring_index > 3 * nside - 1) {
        nphi = 4 * (total_rings - ring_index);
        // Compute lower pixel offset
        int reverse_ring_index = total_rings - ring_index;
        offset_ring = total_pixels - (reverse_ring_index * (reverse_ring_index + 1) * 2);
        pos = -1;
    }
    // Equatorial ring
    else {
        nphi = 4 * nside;
        offset_ring = upper_pixels + (ring_index - nside + 1) * 4 * nside;
        pos = 0;
    }

    int slice_start = offset + (L - nphi / 2);
    int slice_end = offset + nphi;
}

__global__ void spectral_extension(int* data, int* output, int nside, int L, int equatorial_offset_start,
                                   int equatorial_offset_end) {
    // few inits
    int polar_rings = nside - 1;
    int equator_rings = 3 * nside + 1;
    int total_rings = 4 * nside - 1;
    int ftm_size = 2 * L;
    // Compute number of pixels
    int total_pixels = 12 * nside * nside;
    int upper_pixels = nside * (nside - 1) * 2;
    int equator_pixels = 4 * nside * equator_rings;
    // Which ring are we working on
    int current_indx = blockIdx.x * blockDim.x + threadIdx.x;
    int pos(0);
    int indx(0);
    // Compute nphi of current ring
    int nphi(0);

    // ring index
    int ring_index = current_indx / (2 * L);
    // offset for the FTM slice
    int offset = current_indx % (2 * L);
    // offset for original healpix ring
    // Sum of all elements from 0 to n is  n * (n + 1) / 2 in o(1) time .. times 4 to get the number of
    // elements before current ring
    int offset_ring(0);

    // Upper Polar rings
    if (ring_index < nside - 1) {
        nphi = 4 * (ring_index + 1);
        offset_ring = ring_index * (ring_index + 1) * 2;
        pos = 1;
    }
    // Lower Polar rings
    else if (ring_index > 3 * nside - 1) {
        nphi = 4 * (total_rings - ring_index);
        // Compute lower pixel offset
        int reverse_ring_index = total_rings - ring_index;
        offset_ring = total_pixels - (reverse_ring_index * (reverse_ring_index + 1) * 2);
        pos = -1;
    }
    // Equatorial ring
    else {
        nphi = 4 * nside;
        offset_ring = upper_pixels + (ring_index - nside + 1) * 4 * nside;
        pos = 0;
    }

    // Spectral extension
    // The resulting array has size 2 * L and it has these indices :

    // fm[-jnp.arange(L - nphi // 2, 0, -1) % nphi],
    // fm,
    // fm[jnp.arange(L - (nphi + 1) // 2) % nphi],

    // Compute the negative part of the spectrum
    // printf("Offset %d (L + nphi / 2) %d \n", offset, (L + nphi / 2));
    int ring_print = 12;
    if (offset < L - nphi / 2) {
        indx = (-(L - nphi / 2 - offset)) % nphi;
        indx = indx < 0 ? nphi + indx : indx;
        indx = indx + offset_ring;
        output[current_indx] = data[indx];
        if (ring_index == ring_print)
            printf("Negative part: current thread %d, offset %d, ring index %d, ring offset %d, nphi %d , "
                   "target index %d, input element %d output element %d\n",
                   current_indx, offset, ring_index, offset_ring, nphi, indx, data[indx],
                   output[current_indx]);
    }

    // Compute the central part of the spectrum
    else if (offset >= L - nphi / 2 && offset < L + nphi / 2) {
        int center_offset = offset - /*negative part offset*/ (L - nphi / 2);
        indx = center_offset + offset_ring;
        output[current_indx] = data[indx];
        if (ring_index == ring_print)
            printf("Center part: current thread %d, offset %d, center offset %d, ring index %d, ring offset "
                   "%d, nphi %d "
                   "target index %d, input element %d output element %d\n",
                   current_indx, offset, center_offset, ring_index, offset_ring, nphi, indx, data[indx],
                   output[current_indx]);
    }
    // Compute the positive part of the spectrum
    else {
        int reverse_offset = ftm_size - offset;
        indx = (L - (int)((nphi + 1) / 2) - reverse_offset) % nphi;
        indx = indx < 0 ? nphi + indx : indx;
        indx = indx + offset_ring;
        output[current_indx] = data[indx];
        if (ring_index == ring_print)
            printf("Positive part: current thread %d, offset %d, ring index %d, ring offset "
                   "%d, nphi %d  "
                   "target index %d, input element %d output element %d\n",
                   current_indx, offset, ring_index, offset_ring, nphi, indx, data[indx],
                   output[current_indx]);
        // printf("Positive part: element at offset %d came from %d\n", current_indx, indx);
    }

    // Only use global memory for now
    // printf("For current index %d,data index %d ring index is %d and nphi is %d and pos is %d, output is
    // [%d] "
    //        "and original is [%d]\n",
    //        current_indx, indx, ring_index, nphi, pos, output[current_indx], data[indx]);
    //}
}

HRESULT launch_spectral_folding(int* data, int* output, const int& nside, const int& L,
                                const int64& equatorial_offset_start, const int64& equatorial_offset_end,
                                hipStream_t stream) {
    return E_NOTIMPL;
}

HRESULT launch_spectral_extension(int* data, int* output, const int& nside, const int& L,
                                  const int64& equatorial_offset_start, const int64& equatorial_offset_end,
                                  hipStream_t stream) {
    // Launch the kernel
    std::cout << "Launching kernel" << std::endl;
    int block_size = 128;
    int ftm_elements = 2 * L * (4 * nside - 1);
    int grid_size = (ftm_elements + block_size - 1) / block_size;
    std::cout << "Grid size: " << grid_size << std::endl;
    std::cout << "Block size: " << block_size << std::endl;
    std::cout << "L: " << L << std::endl;
    std::cout << "equatorial_offset_start: " << equatorial_offset_start << std::endl;
    std::cout << "equatorial_offset_end: " << equatorial_offset_end << std::endl;

    spectral_extension<<<grid_size, block_size, 0, stream>>>(data, output, nside, L, equatorial_offset_start,
                                                             equatorial_offset_end);

    checkCudaErrors(hipGetLastError());
    return S_OK;
}

}  // namespace s2fftKernels