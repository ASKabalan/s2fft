#include "hip/hip_runtime.h"
#include <algorithm>
#include "s2fft.h"
#include <array>
#include <iostream>
#include <ostream>
#include <sstream>
#include "hipfft/hipfft.h"
#include "perfostep.hpp"
#include "s2fft_kernels.h"

using namespace s2fft;

using Type = hipComplex;

int main() {
    Perfostep perfostep;
    int nside = 4;
    int L = 2 * nside;
    int total_pixels = 12 * nside * nside;

    // Compute the flm size
    int polar_pixels = 4 * nside * (nside - 1);
    int equator_rings_num = (total_pixels - polar_pixels) / (4 * nside);
    int num_rings = equator_rings_num + 2 * (nside - 1);
    int flm_size = num_rings * (4 * nside);

    std::cout << "nside: " << nside << std::endl;
    std::cout << "Total pixels: " << total_pixels << std::endl;

    hipComplex *h_vec_in = new hipComplex[total_pixels];
    hipComplex *h_vec_out = new hipComplex[flm_size];
    hipComplex *d_vec_in;
    hipComplex *d_vec_out;

    hipMallocManaged(&d_vec_in, total_pixels * sizeof(hipComplex));
    hipMallocManaged(&d_vec_out, flm_size * sizeof(hipComplex));

    // Initialize host vectors using std::generate
    int start_index(0);
    std::generate(h_vec_in, h_vec_in + total_pixels, [&start_index]() {
        hipComplex c;
        c.x = start_index;
        c.y = 0.0f;
        start_index += 1;
        return c;
    });

    // Generate sequence

    // Copy host data to device
    hipMemcpy(d_vec_in, h_vec_in, total_pixels * sizeof(hipComplex), hipMemcpyHostToDevice);

    ////// Create cudastream
    hipStream_t stream;
    hipStreamCreate(&stream);
    

    s2fftKernels::launch_spectral_extension(d_vec_in, d_vec_out, nside, L, total_pixels, stream);
    hipDeviceSynchronize();
    //std::cout << "Original" << std::endl;
    //for (int i = 0; i < total_pixels; i++) {
    //    std::cout << "[" << i << "] " << h_vec_in[i].x << " + " << h_vec_in[i].y << "i" << std::endl;
//
    //    //if (i == 3) break;
    //}

   //s2fftDescriptor desc(nside, L, true, true, fft_norm::BACKWARD, true);
   //s2fftExec exec;
   //size_t worksize(0);
   //perfostep.Start("Initialize");
   //exec.Initialize(desc, worksize);
   //perfostep.Stop();
   //std::cout << "worksize: " << worksize << std::endl;

   ////


   ////// Set first buffer to the data
   //void **buffers = (void **)malloc(2 * sizeof(void *));
   //buffers[0] = d_vec;
   //worksize = total_pixels * sizeof(hipComplex);
   //buffers[1] = (void *)worksize;
   //buffers[2] = d_vec_out;

   ////// ********************************************************
   ////// Perform forward
   ////// ********************************************************
   //perfostep.Start("Forward");
   //exec.Forward(desc, stream, buffers);
   //hipStreamSynchronize(stream);
   //hipDeviceSynchronize();
   //perfostep.Stop();
   //std::cout << "Executed Forward" << std::endl;

   //// ********************************************************
   //// Perform Backward
   //// ********************************************************
   //perfostep.Start("Backward");
   //exec.Backward(desc, stream, buffers);
   //hipStreamSynchronize(stream);
   //perfostep.Stop();
   //std::cout << "Executed Backward" << std::endl;

   //perfostep.Report();

   //// Copy device data to host
   hipMemcpy(h_vec_out, d_vec_out, flm_size * sizeof(hipComplex), hipMemcpyDeviceToHost);

   std::cout << "Output" << std::endl;
   for (int i = 0; i < flm_size; i++) {
       std::cout << "[" << i << "] " << h_vec_out[i].x << " + " << h_vec_out[i].y << "i" << std::endl;

       if (i == 3) break;
   }

   //// Check Maximum reconstruction error
   //float max_error = 0.0f;
   //for (int i = 0; i < total_pixels; i++) {
   //    float error =
   //            std::max(std::abs(h_vec_in[i].x - h_vec_out[i].x), std::abs(h_vec_in[i].y - h_vec_out[i].y));
   //    if (error > 0.5f) {
   //        // std::cout << "Element: " << i << " Error: " << error << std::endl;
   //    }
   //    max_error = std::max(max_error, error);
   //}
   //std::cout << "Max error: " << max_error << std::endl;

    // Free memory
    delete[] h_vec_in;
    delete[] h_vec_out;
    hipFree(d_vec_in);
    hipFree(d_vec_out);

    std::cout << "Done" << std::endl;
    return 0;
}
