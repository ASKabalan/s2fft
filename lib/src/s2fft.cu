#include "hip/hip_runtime.h"
#include "s2fft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "hresult.h"
#include <cmath>  // has to be included before cuda/std/complex
#include <cstddef>
#include <cuda/std/complex>
#include <algorithm>
#include <iostream>
#include <iostream>
#include <iterator>
#include <numeric>

#include <vector>
#include "s2fft_callbacks.cuh"

namespace s2fft {

template <typename real_t>
HRESULT s2fftExec<real_t>::Initialize(const s2fftDescriptor &descriptor, size_t &worksize) {
    m_nside = descriptor.nside;
    m_total_pixels = 12 * m_nside * m_nside;

    size_t start_index(0);
    size_t end_index(12 * m_nside * m_nside);
    size_t nphi(0);
    const bool &shift = descriptor.shift;
    m_upper_ring_offsets.reserve(m_nside - 1);
    m_lower_ring_offsets.reserve(m_nside - 1);

    for (size_t i = 0; i < m_nside - 1; i++) {
        nphi = 4 * (i + 1);
        m_upper_ring_offsets.push_back(start_index);
        m_lower_ring_offsets.push_back(end_index - nphi);

        start_index += nphi;
        end_index -= nphi;
    }
    m_equatorial_offset_start = start_index;
    m_equatorial_offset_end = end_index;
    m_equatorial_ring_num = (end_index - start_index) / (4 * m_nside);

    // Plan creation
    for (size_t i = 0; i < m_nside - 1; i++) {
        size_t polar_worksize{0};
        int64 upper_ring_offset = m_upper_ring_offsets[i];
        int64 lower_ring_offset = m_lower_ring_offsets[i];

        hipfftHandle plan{};
        hipfftHandle inverse_plan{};
        CUFFT_CALL(hipfftCreate(&plan));
        CUFFT_CALL(hipfftCreate(&inverse_plan));
        // Plans are done on upper and lower polar rings
        int rank = 1;                      // 1D FFT  : In our case the rank is always 1
        int batch_size = 2;                // Number of rings to transform
        int64 n[] = {4 * ((int64)i + 1)};  // Size of each FFT 4 times the ring number (first is 4, second is
                                           // 8, third is 12, etc)
        int64 inembed[] = {0};             // Stride of input data (meaningless but has to be set)
        int64 istride = 1;  // Distance between consecutive elements in the same batch always 1 since we
                            // have contiguous data
        int64 idist = lower_ring_offset -
                      upper_ring_offset;  // Distance between the starting points of two consecutive
                                          // batches, it is equal to the distance between the two rings
        int64 onembed[] = {0};            // Stride of output data (meaningless but has to be set)
        int64 ostride = 1;  // Distance between consecutive elements in the output batch, also 1 since
                            // everything is done in place
        int64 odist =
                lower_ring_offset - upper_ring_offset;  // Same as idist since we want to transform in place

        CUFFT_CALL(hipfftMakePlanMany64(plan, rank, n, inembed, istride, idist, onembed, ostride, odist,
                                       HIPFFT_C2C, batch_size, &polar_worksize));

        CUFFT_CALL(hipfftMakePlanMany64(inverse_plan, rank, n, inembed, istride, idist, onembed, ostride,
                                       odist, HIPFFT_C2C, batch_size, &polar_worksize));

        int64 params[4];
        int64 *params_dev;
        params[0] = n[0];
        params[1] = idist;
        params[2] = upper_ring_offset;
        params[3] = lower_ring_offset;
        hipMalloc(&params_dev, 4 * sizeof(int64));
        hipMemcpy(params_dev, params, 4 * sizeof(int64), hipMemcpyHostToDevice);

        // Backward shift is a load call back .. set it on its own
        if (shift) {
            CUFFT_CALL(hipfftXtSetCallback(inverse_plan, (void **)&s2fftKernels::ifft_shift_ptr,
                                          HIPFFT_CB_LD_COMPLEX, (void **)&params_dev));
        }
        //  Set the callback for the forward and backward ffts
        switch (descriptor.norm) {
            case fft_norm::ORTHO:
                CUFFT_CALL(hipfftXtSetCallback(plan, (void **)&FFT_NORM_ORTHO(shift), HIPFFT_CB_ST_COMPLEX,
                                              (void **)&params_dev));
                // For inverse plans the shifting is done in the load callback
                CUFFT_CALL(hipfftXtSetCallback(inverse_plan, (void **)&FFT_NORM_ORTHO(false),
                                              HIPFFT_CB_ST_COMPLEX, (void **)&params_dev));
                break;
            case fft_norm::BACKWARD:
                // No normalization is done for the forward fft but the shifting is done if requested
                if (shift) {
                    CUFFT_CALL(hipfftXtSetCallback(plan, (void **)&s2fftKernels::fft_shift_ptr,
                                                  HIPFFT_CB_ST_COMPLEX, (void **)&params_dev));
                }
                // For inverse plans the shifting is done in the load callback
                CUFFT_CALL(hipfftXtSetCallback(inverse_plan, (void **)&FFT_NORM(false), HIPFFT_CB_ST_COMPLEX,
                                              (void **)&params_dev));
                break;
            case fft_norm::FORWARD:
                CUFFT_CALL(hipfftXtSetCallback(plan, (void **)&FFT_NORM(shift), HIPFFT_CB_ST_COMPLEX,
                                              (void **)&params_dev));
                break;
            case fft_norm::NONE:
                if (shift) {
                    CUFFT_CALL(hipfftXtSetCallback(plan, (void **)&s2fftKernels::fft_shift_ptr,
                                                  HIPFFT_CB_ST_COMPLEX, (void **)&params_dev));
                }
        }

        m_polar_plans.push_back(plan);
        m_inverse_polar_plans.push_back(inverse_plan);
    }
    // Equator plan

    // Equator is a matrix with size 4 * m_nside x equatorial_ring_num
    // hipfftMakePlan1d is enough for this case

    size_t equator_worksize{0};
    int64 equator_size = (4 * m_nside);
    // Forward plan
    CUFFT_CALL(hipfftCreate(&m_equator_plan));
    CUFFT_CALL(hipfftMakePlanMany64(m_equator_plan, 1, &equator_size, nullptr, 1, 1, nullptr, 1, 1, HIPFFT_C2C,
                                   m_equatorial_ring_num, &equator_worksize));
    // Inverse plan
    CUFFT_CALL(hipfftCreate(&m_inverse_equator_plan));
    CUFFT_CALL(hipfftMakePlanMany64(m_inverse_equator_plan, 1, &equator_size, nullptr, 1, 1, nullptr, 1, 1,
                                   HIPFFT_C2C, m_equatorial_ring_num, &equator_worksize));

    int64 equator_params[3];
    equator_params[0] = equator_size;
    equator_params[1] = m_equatorial_offset_start;
    // Dummy param, the offset of any equator element is less than the offset of the last ring
    equator_params[2] = m_equatorial_offset_end;
    int64 *equator_params_dev;
    hipMalloc(&equator_params_dev, 3 * sizeof(int64));
    hipMemcpy(equator_params_dev, equator_params, 3 * sizeof(int64), hipMemcpyHostToDevice);

    if (shift) {
        CUFFT_CALL(hipfftXtSetCallback(m_inverse_equator_plan, (void **)&s2fftKernels::ifft_shift_eq_ptr,
                                      HIPFFT_CB_LD_COMPLEX, (void **)&equator_params_dev));
    }

    switch (descriptor.norm) {
        case fft_norm::ORTHO:
            CUFFT_CALL(hipfftXtSetCallback(m_equator_plan, (void **)&FFT_NORM_ORTHO_EQ(shift),
                                          HIPFFT_CB_ST_COMPLEX, (void **)&equator_params_dev));
            CUFFT_CALL(hipfftXtSetCallback(m_inverse_equator_plan, (void **)&FFT_NORM_ORTHO_EQ(false),
                                          HIPFFT_CB_ST_COMPLEX, (void **)&equator_params_dev));
            break;
        case fft_norm::BACKWARD:
            // No normalization is done for the forward fft but the shifting is done if requested
            if (shift)
                CUFFT_CALL(hipfftXtSetCallback(m_equator_plan, (void **)&s2fftKernels::fft_shift_eq_ptr,
                                              HIPFFT_CB_ST_COMPLEX, (void **)&equator_params_dev));
            // For inverse plans the shifting is done in the load callback
            CUFFT_CALL(hipfftXtSetCallback(m_inverse_equator_plan, (void **)&FFT_NORM_EQ(false),
                                          HIPFFT_CB_ST_COMPLEX, (void **)&equator_params_dev));
            break;
        case fft_norm::FORWARD:
            CUFFT_CALL(hipfftXtSetCallback(m_equator_plan, (void **)&FFT_NORM_EQ(shift), HIPFFT_CB_ST_COMPLEX,
                                          (void **)&equator_params_dev));
            break;
        case fft_norm::NONE:
            if (shift) {
                CUFFT_CALL(hipfftXtSetCallback(m_equator_plan, (void **)&s2fftKernels::fft_shift_eq_ptr,
                                              HIPFFT_CB_ST_COMPLEX, (void **)&equator_params_dev));
            }
    }

    return S_OK;
}

template <typename real_t>
HRESULT s2fftExec<real_t>::Forward(const s2fftDescriptor &desc, hipStream_t stream, void *data) {
    // Polar rings ffts*/
    complex_t *data_c = reinterpret_cast<complex_t *>(data);
    for (int i = 0; i < m_nside - 1; i++) {
        int upper_ring_offset = m_upper_ring_offsets[i];

        CUFFT_CALL(hipfftSetStream(m_polar_plans[i], stream))
        CUFFT_CALL(hipfftXtExec(m_polar_plans[i], data_c + upper_ring_offset, data_c + upper_ring_offset,
                               HIPFFT_FORWARD));
    }

    // Equator fft
    CUFFT_CALL(hipfftSetStream(m_equator_plan, stream))
    CUFFT_CALL(hipfftXtExec(m_equator_plan, data_c + m_equatorial_offset_start,
                           data_c + m_equatorial_offset_start, HIPFFT_FORWARD));
    //
    return S_OK;
}

template <typename real_t>
HRESULT s2fftExec<real_t>::Backward(const s2fftDescriptor &desc, hipStream_t stream, void *data) {
    // Polar rings inverse FFTs
    for (int i = 0; i < m_nside - 1; i++) {
        int upper_ring_offset = m_upper_ring_offsets[i];

        CUFFT_CALL(hipfftSetStream(m_inverse_polar_plans[i], stream))
        CUFFT_CALL(hipfftXtExec(m_inverse_polar_plans[i], data + upper_ring_offset, data + upper_ring_offset,
                               HIPFFT_BACKWARD));
    }
    // Equator inverse FFT
    CUFFT_CALL(hipfftSetStream(m_inverse_equator_plan, stream))
    CUFFT_CALL(hipfftXtExec(m_inverse_equator_plan, data + m_equatorial_offset_start,
                           data + m_equatorial_offset_start, HIPFFT_BACKWARD));
    //
    return S_OK;
}

template class s2fftExec<float>;
template class s2fftExec<double>;

}  // namespace s2fft