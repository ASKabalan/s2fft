#include "s2fft.h"
#include "hresult.h"
#include <cmath>  // has to be included before cuda/std/complex
#include <cstddef>
#include <cuda/std/complex>
#include <algorithm>
#include <iostream>
#include <iostream>
#include <iterator>
#include <numeric>
#include <vector>

#include <vector>
#include "hipfft/hipfft.h"
#include <hipfft/hipfftXt.h>

namespace s2fft {

__device__ hipfftComplex backward_norm_and_fftshift(void *dataIn, size_t offset, void *callerInfo,
                                                   void *sharedPtr) {
    hipfftComplex *data = (hipfftComplex *)dataIn;
    int size = *((int *)callerInfo);  // Assuming callerInfo holds the size of the FFT

    // Apply backward normalization
    float norm_factor = 1.0f / size;
    data[offset].x *= norm_factor;
    data[offset].y *= norm_factor;

    // Apply FFT shift
    int half_size = size / 2;
    int shifted_index = (offset + half_size) % size;
    hipfftComplex temp = data[offset];
    data[offset] = data[shifted_index];
    data[shifted_index] = temp;

    return data[offset];
}



HRESULT s2fftExec::Initialize(const s2fftDescriptor &descriptor, size_t &worksize) {
    m_nside = descriptor.nside;
    m_total_pixels = 12 * m_nside * m_nside;

    int start_index(0);
    int end_index(12 * m_nside * m_nside);
    int nphi(0);
    m_upper_ring_offsets.reserve(m_nside - 1);
    m_lower_ring_offsets.reserve(m_nside - 1);

    for (int i = 0; i < m_nside - 1; i++) {
        nphi = 4 * (i + 1);
        m_upper_ring_offsets.push_back(start_index);
        m_lower_ring_offsets.push_back(end_index - nphi);

        start_index += nphi;
        end_index -= nphi;
    }
    equatorial_offset = start_index;
    equatorial_ring_num = (end_index - start_index) / (4 * m_nside);

    // Initialize cufftCallbacks
    hipfftCallbackStoreC callbackPtr;
    hipError_t err = hipMemcpyFromSymbol(&callbackPtr, HIP_SYMBOL(backward_norm_and_fftshift), sizeof(callbackPtr));
    if (err != hipSuccess) {
        // Handle error
    }

    // Plan creation
    for (int i = 0; i < m_nside - 1; i++) {
        size_t polar_worksize{0};
        int upper_ring_offset = m_upper_ring_offsets[i];
        int lower_ring_offset = m_lower_ring_offsets[i];

        hipfftHandle plan{};
        hipfftHandle inverse_plan{};
        CUFFT_CALL(hipfftCreate(&plan));
        CUFFT_CALL(hipfftCreate(&inverse_plan));
        // Plans are done on upper and lower polar rings
        int rank = 1;             // 1D FFT  : In our case the rank is always 1
        int batch_size = 2;       // Number of rings to transform
        int n[] = {4 * (i + 1)};  // Size of each FFT 4 times the ring number (first is 4, second is 8, third is 12, etc)
        int inembed[] = {0};      // Stride of input data (meaningless but has to be set)
        int istride = 1;          // Distance between consecutive elements in the same batch always 1 since we have contiguous data
        int idist = lower_ring_offset -
                    upper_ring_offset;  // Distance between the starting points of two consecutive batches, it is equal to the distance between the two rings
        int onembed[] = {0};            // Stride of output data (meaningless but has to be set)
        int ostride = 1;                // Distance between consecutive elements in the output batch, also 1 since everything is done in place
        int odist = lower_ring_offset - upper_ring_offset; // Same as idist since we want to transform in place

        CUFFT_CALL(hipfftMakePlanMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist,
                                     HIPFFT_C2C, batch_size, &polar_worksize));
        CUFFT_CALL(
                hipfftXtSetCallback(m_polar_plans[i], (void **)&callbackPtr, HIPFFT_CB_LD_COMPLEX, (void **)n));

        m_polar_plans.push_back(plan);
    }
    // Equator plan

    // Equator is a matrix with size 4 * m_nside x equatorial_ring_num
    // hipfftMakePlan1d is enough for this case

    size_t equator_worksize{0};
    CUFFT_CALL(hipfftCreate(&m_equator_plan));
    CUFFT_CALL(hipfftMakePlan1d(m_equator_plan, (4 * m_nside), HIPFFT_C2C, equatorial_ring_num,
                               &equator_worksize));

    return S_OK;
}

HRESULT s2fftExec::Forward(const s2fftDescriptor &desc, hipStream_t stream, void **buffers) {
    void *data_d = buffers[0];
    hipfftComplex *data_c_d = static_cast<hipfftComplex *>(data_d);
    // void *work_d = buffers[1];
    // void *data_out_d = buffers[2];
    // hipfftComplex *data_c_o = static_cast<hipfftComplex *>(data_out_d);

    // Polar rings ffts
    std::cout << "number of plans: " << m_polar_plans.size() << std::endl;
    for (int i = 0; i < m_nside - 1; i++) {
        int upper_ring_offset = m_upper_ring_offsets[i];

        CUFFT_CALL(hipfftSetStream(m_polar_plans[i], stream))
        CUFFT_CALL(hipfftExecC2C(m_polar_plans[i], data_c_d + upper_ring_offset, data_c_d + upper_ring_offset,
                                HIPFFT_FORWARD));
    }

    // Equator fft
    CUFFT_CALL(hipfftSetStream(m_equator_plan, stream))
    CUFFT_CALL(hipfftExecC2C(m_equator_plan, data_c_d + equatorial_offset, data_c_d + equatorial_offset,
                            HIPFFT_FORWARD));

    return S_OK;
}

HRESULT s2fftExec::Backward(const s2fftDescriptor &desc, hipStream_t stream, void **buffers) {
    void *data_d = buffers[0];
    hipfftComplex *data_c_d = static_cast<hipfftComplex *>(data_d);

    // Polar rings inverse FFTs
    for (int i = 0; i < m_nside - 1; i++) {
        int upper_ring_offset = m_upper_ring_offsets[i];

        CUFFT_CALL(hipfftSetStream(m_polar_plans[i], stream))
        CUFFT_CALL(hipfftExecC2C(m_polar_plans[i], data_c_d + upper_ring_offset, data_c_d + upper_ring_offset,
                                HIPFFT_BACKWARD));
    }

    // Equator inverse FFT
    CUFFT_CALL(hipfftSetStream(m_equator_plan, stream))
    CUFFT_CALL(hipfftExecC2C(m_equator_plan, data_c_d + equatorial_offset, data_c_d + equatorial_offset,
                            HIPFFT_BACKWARD));

    return S_OK;
}
}  // namespace s2fft